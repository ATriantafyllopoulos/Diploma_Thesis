#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>
/*#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"*/
#include <stdio.h>

hipError_t initializeWithCuda(struct hipGraphicsResource* testingVBO_CUDA, size_t *num_bytes);

__global__ void initializeKernel(float3* positions)
{
	int i = threadIdx.y * blockDim.x + threadIdx.x;
	int j = threadIdx.y;
	positions[i].x = (float)i;
	positions[i].y = (float)j;
	positions[i].z = -10.0;
}

// Helper function for using CUDA to initialize particle positions.
hipError_t initializeWithCuda(struct hipGraphicsResource* testingVBO_CUDA, size_t *num_bytes)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	float3* positions;
	
	cudaStatus = hipGraphicsMapResources(1, &testingVBO_CUDA, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsMapResources returned error code %d before launching initializeKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&positions, num_bytes, testingVBO_CUDA);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsResourceGetMappedPointer returned error code %d before launching initializeKernel!\n", cudaStatus);
		goto Error;
	}
	

	// Launch a kernel on the GPU with one thread for each element.
	initializeKernel << <2, 512 >> >(positions);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initializeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching initializeKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGraphicsUnmapResources(1, &testingVBO_CUDA, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsUnmapResources returned error code %d after launching initializeKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}
