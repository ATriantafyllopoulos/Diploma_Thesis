#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void animateKernel(float3* positions, float offset)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	positions[index].x += offset;
}

__global__ void initializeKernel(float3* positions)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x;
	int j = blockIdx.x;
	positions[index].x = (float)i;
	positions[index].y = (float)j;
	positions[index].z = -10.0;
}

__global__ void meshCreationKernel(float3 *positions, hipPitchedPtr gridCoordinates, float3 smallestCoords, float d)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float4 *g = (float4 *)gridCoordinates.ptr;
	size_t    pitch = gridCoordinates.pitch;
	size_t    slicePitch = pitch * 10;

	int xPos = (positions[index].x - smallestCoords.x) / d;
	int yPos = (positions[index].y - smallestCoords.y) / d;
	int zPos = (positions[index].z - smallestCoords.z) / d;

	//zPos * 10 * 10 + yPos * 10 + xPos
	int gridIndex = zPos * slicePitch + yPos * pitch + xPos;
	
	g[gridIndex].x = index;
}


void dummyInitialization(float3* positions, const int &numberOfParticles)
{
	int numOfThreads = 512;
	initializeKernel << <(numberOfParticles + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "CUDA engine failed!" << std::endl;
		std::cout << "callback function: dummyInitialization" << std::endl;
		std::cout << "Error type: " << cudaStatus << std::endl;
		system("pause"); //for now pause system when an error occurs (only for debug purposes)
	}
}

void dummyAnimation(float3* positions, const double &offset, const int &numberOfParticles)
{
	int numOfThreads = 512;
	animateKernel << <(numberOfParticles + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, offset);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "CUDA engine failed!" << std::endl;
		std::cout << "callback function: dummyAnimation" << std::endl;
		std::cout << "Error type: " << cudaStatus << std::endl;
		system("pause"); //for now pause system when an error occurs (only for debug purposes)
	}
}

void dummyMeshCreation(float3 *positions, hipPitchedPtr gridCoordinates, float3 smallestCoords, const float &d, const int &numberOfParticles)
{
	int numOfThreads = 512;
	meshCreationKernel << <(numberOfParticles + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, gridCoordinates, smallestCoords, d);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "CUDA engine failed!" << std::endl;
		std::cout << "callback function: dummyMeshCreation" << std::endl;
		std::cout << "Error type: " << cudaStatus << std::endl;
		system("pause"); //for now pause system when an error occurs (only for debug purposes)
	}
}

