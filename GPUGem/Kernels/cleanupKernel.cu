#include "hip/hip_runtime.h"


/*
Using void argument to make this function compatible with all types of pointers.
*/
hipError_t cleanup(void** pt)
{
	hipError_t cudaStatus = hipFree(*pt);
	return cudaStatus;
}