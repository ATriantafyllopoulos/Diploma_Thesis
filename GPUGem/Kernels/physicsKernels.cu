#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
hipError_t cudaFail(hipError_t cudaStatus, char *funcName);

__global__ void animateKernel(float3* positions, float offset)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	positions[index].x += offset;
}

__global__ void initializeKernel(float3* positions, float3* linearMomenta)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x;
	int j = blockIdx.x;

	positions[index] = make_float3((float)i, (float)j, -10.f);

	//particles are initially static
	linearMomenta[index] = make_float3(0.f, 0.f, 0.f);
}

hipError_t dummyInitialization(float3* positions, float3* linearMomenta, const int &numberOfPrimitives)
{
	int numOfThreads = 512;
	hipError_t cudaStatus = hipMalloc((void**)&linearMomenta, numberOfPrimitives * sizeof(float3));
	if (cudaStatus != hipSuccess)
	{
		hipFree(linearMomenta);
		return cudaFail(cudaStatus, "dummyInitialization_malloc");
	}
	initializeKernel << <(numberOfPrimitives + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, linearMomenta);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		hipFree(linearMomenta);
		return cudaFail(cudaStatus, "dummyInitialization");
	}
	return hipSuccess;
}

hipError_t dummyAnimation(float3* positions, const double &offset, const int &numberOfPrimitives)
{
	int numOfThreads = 512;
	animateKernel << <(numberOfPrimitives + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, offset);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return cudaFail(cudaStatus, "dummyAnimation");
	//std::cout << "CUDA animation was successful." << std::endl;
	return hipSuccess;
}

hipError_t cudaFail(hipError_t cudaStatus, char *funcName)
{
	std::cout << "CUDA engine failed!" << std::endl;
	std::cout << "callback function:" << funcName << std::endl;
	std::cout << "Error code: " << cudaStatus << std::endl;
	std::cout << "Error type: " << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "Enter random character to continue..." << std::endl;
	int x;
	std::cin >> x;
	exit(1);
	return cudaStatus;
}