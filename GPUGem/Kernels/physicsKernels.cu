#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
hipError_t cudaFail(hipError_t cudaStatus, char *funcName);

__global__ void animateKernel(float3* positions, float offset)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	positions[index].x += offset;
}

__global__ void initializeKernel(float3* positions, float3* linearMomenta)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x;
	int j = blockIdx.x;

	positions[index] = make_float3((float)i, (float)j, -10.f);

	//particles are initially static
	linearMomenta[index] = make_float3(0.f, 0.f, 0.f);
}

__global__ void meshCreationKernel(float3 *positions, hipPitchedPtr gridCoordinates, float3 smallestCoords, float d)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	float4 *g = (float4 *)gridCoordinates.ptr;
	size_t    pitch = gridCoordinates.pitch;
	size_t    slicePitch = pitch * 10;

	int xPos = (positions[index].x - smallestCoords.x) / d;
	int yPos = (positions[index].y - smallestCoords.y) / d;
	int zPos = (positions[index].z - smallestCoords.z) / d;

	//zPos * 10 * 10 + yPos * 10 + xPos
	int gridIndex = zPos * slicePitch + yPos * pitch + xPos;
	
	g[gridIndex].x = index;
}


hipError_t dummyInitialization(float3* positions, float3* linearMomenta, const int &numberOfPrimitives)
{
	int numOfThreads = 512;
	hipError_t cudaStatus = hipMalloc((void**)&linearMomenta, numberOfPrimitives * sizeof(float3));
	if (cudaStatus != hipSuccess)
	{
		hipFree(linearMomenta);
		return cudaFail(cudaStatus, "dummyInitialization_malloc");
	}
	initializeKernel << <(numberOfPrimitives + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, linearMomenta);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		hipFree(linearMomenta);
		return cudaFail(cudaStatus, "dummyInitialization");
	}
	return hipSuccess;
}

hipError_t dummyAnimation(float3* positions, const double &offset, const int &numberOfPrimitives)
{
	int numOfThreads = 512;
	animateKernel << <(numberOfPrimitives + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, offset);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return cudaFail(cudaStatus, "dummyAnimation");
	//std::cout << "CUDA animation was successful." << std::endl;
	return hipSuccess;
}

hipError_t dummyMeshCreation(float3 *positions, hipPitchedPtr gridCoordinates, float3 smallestCoords, const float &d, const int &numberOfPrimitives)
{
	int numOfThreads = 512;
	meshCreationKernel << <(numberOfPrimitives + numOfThreads - 1) / numOfThreads, numOfThreads >> >(positions, gridCoordinates, smallestCoords, d);
	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		return cudaFail(cudaStatus, "dummyMeshCreation");
	return hipSuccess;
}

hipError_t cudaFail(hipError_t cudaStatus, char *funcName)
{
	std::cout << "CUDA engine failed!" << std::endl;
	std::cout << "callback function:" << funcName << std::endl;
	std::cout << "Error code: " << cudaStatus << std::endl;
	std::cout << "Error type: " << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "Enter random character to continue..." << std::endl;
	int x;
	std::cin >> x;
	exit(1);
	return cudaStatus;
}