#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Primitives.h"
#include <algorithm>
#include <iostream>
#define HIPCUB_STDERR

//define _HipcubLog to avoid encountering error: "undefined reference"
#if !defined(_HipcubLog)
#if (HIPCUB_ARCH == 0)
#define _HipcubLog(format, ...) printf(format,__VA_ARGS__);
#elif (HIPCUB_ARCH >= 200)
#define _HipcubLog(format, ...) printf("[block (%d,%d,%d), thread (%d,%d,%d)]: " format, blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x, __VA_ARGS__);
#endif
#endif

//cub headers
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/test/test_util.h>

hipError_t update(Primitive* leafNodes, const float &timeStep, const int &numberOfPrimitives, const int &numberOfThreads);
/*
This particular file is a mess. I definitely need to clean it up on the weekend.
It's decided. The weekend will be dedicated to cleaning up the code.
This will need to be part of a class.
*/
__global__ void handleCollisions(Primitive *leafNodes, int numberOfPrimitives);

hipError_t cudaFail(hipError_t cudaStatus, char *funcName);

hipError_t generateHierarchy(Primitive *internalNodes,
	Primitive* leafNodes,
	unsigned int* sortedMortonCodes,
	int           numberOfPrimitives);

__device__ inline float MIN(float x, float y)
{
	return x < y ? x : y;
}

__device__ inline float MAX(float x, float y)
{
	return x > y ? x : y;
}

//Expands a 10-bit integer into 30 bits
//by inserting 2 zeros after each bit.
__device__ unsigned int expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int morton3D(float x, float y, float z)
{
	x = MIN(MAX(x * 1024.0f, 0.0f), 1023.0f);
	y = MIN(MAX(y * 1024.0f, 0.0f), 1023.0f);
	z = MIN(MAX(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = expandBits((unsigned int)x);
	unsigned int yy = expandBits((unsigned int)y);
	unsigned int zz = expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}

__device__ inline bool checkOverlap(Primitive *query, Primitive *node)
{
	float dist = __fsqrt_rd((node->centroid.x - query->centroid.x) * (node->centroid.x - query->centroid.x) +
		(node->centroid.y - query->centroid.y) * (node->centroid.y - query->centroid.y) +
		(node->centroid.z - query->centroid.z) * (node->centroid.z - query->centroid.z));
	return dist < node->radius + query->radius;
}

__device__ void traverseIterative(Primitive *root,
	Primitive* queryLeaf)
{
	// Allocate traversal stack from thread-local memory,
	// and push NULL to indicate that there are no postponed nodes.
	Primitive* stack[64]; //AT: Is 64 the correct size to use?
	Primitive** stackPtr = stack;
	//when stack is empty thread will return
	*stackPtr++ = NULL; // push NULL at beginning

	// Traverse nodes starting from the root.
	Primitive* node = root;
	do
	{
		// Check each child node for overlap.
		Primitive* childL = node->left;
		Primitive* childR = node->right;
		bool overlapL = (checkOverlap(queryLeaf, childL));
		bool overlapR = (checkOverlap(queryLeaf, childR));

		if (node->leftmost <= queryLeaf->id)
			overlapL = false;

		if (node->rightmost <= queryLeaf->id)
			overlapR = false;
		
		// Query overlaps a leaf node => report collision.
		if (overlapL && childL->isLeaf)
			queryLeaf->collisions[queryLeaf->collisionCounter++] = childL;
		queryLeaf->collisionCounter = queryLeaf->collisionCounter > 7 ? 0 : queryLeaf->collisionCounter; //avoid overflow
		
		if (overlapR && childR->isLeaf)
			queryLeaf->collisions[queryLeaf->collisionCounter++] = childR;
		queryLeaf->collisionCounter = queryLeaf->collisionCounter > 7 ? 0 : queryLeaf->collisionCounter; //avoid overflow
		
		// Query overlaps an internal node => traverse.
		bool traverseL = (overlapL && !childL->isLeaf);
		bool traverseR = (overlapR && !childR->isLeaf);

		if (!traverseL && !traverseR)
			node = *--stackPtr; // pop
		else
		{
			node = (traverseL) ? childL : childR;
			if (traverseL && traverseR)
				*stackPtr++ = childR; // push
		}
	} while (node != NULL);
}

__global__ void generateMortonCodes(float3 *positions, unsigned int *mortonCodes, int numberOfPrimitives)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= numberOfPrimitives)
		return;

	mortonCodes[index] = morton3D(positions[index].x, positions[index].y, positions[index].z);
}

__global__ void findPotentialCollisions(Primitive *internalNodes, Primitive *leafNodes, int numOfLeaves)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= numOfLeaves)
		return;

	Primitive *leaf = leafNodes + index;
	traverseIterative(internalNodes, leaf);
}

/*
routine is called before BVH is created
leaf node primitives are yet unsorted
radius and mass are currently hard-coded -> 1 [OPEN]
Make radius parametric. Design interface to input parameters. [OPEN]
*/
__global__ void constructLeafNodes(Primitive* leafNodes,
	float3 *positions,
	float *linearMomenta,
	int numberOfPrimitives)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numberOfPrimitives)
		return;
	leafNodes[index].id = index; //leaf nodes are unsorted
	leafNodes[index].isLeaf = true;
	//each leaf reports rightmost leaf of left and right sutree as itself
	leafNodes[index].leftmost = index;
	leafNodes[index].rightmost = index;

	leafNodes[index].left = NULL;
	leafNodes[index].right = NULL;

	leafNodes[index].parent = NULL;

	//copying state vectors
	//too slow because of too many memory calls
	leafNodes[index].centroid = positions[index];
	leafNodes[index].radius = 1;
	leafNodes[index].mass = 1;
	leafNodes[index].collisionCounter = 0;

	//float3 a = linearMomenta[index];
	//leafNodes[index].linearMomentum = linearMomenta[index];
	//leafNodes[index].linearMomentum = *(float3*)((double*)(linearMomenta + index));
	/*float x = *(float*)(linearMomenta + index);
	float y = *(float*)(linearMomenta + index + 32);
	float z = *(float*)(linearMomenta + index + 64);

	leafNodes[index].linearMomentum = make_float3(x, y, z);*/
	int idx = 3 * blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float sMomenta[512 * 3];
	sMomenta[threadIdx.x] = linearMomenta[idx];
	sMomenta[threadIdx.x + 512] = linearMomenta[idx + 512];
	sMomenta[threadIdx.x + 1024] = linearMomenta[idx + 1024];
	__syncthreads();

	leafNodes[idx].linearMomentum.x = (sMomenta)[threadIdx.x];
	leafNodes[idx + 512].linearMomentum.y = (sMomenta)[threadIdx.x + 512];
	leafNodes[idx + 1024].linearMomentum.z = (sMomenta)[threadIdx.x + 1024];
	//leafNodes[index].angularMomentum = angularMomentums[index];
	//leafNodes[index].quaternion = quaternions[index];
}

/*
Potential error: sorting the leaf nodes by value, using Morton codes as keys, is not done on the code I found. [OPEN]
*/
hipError_t detectCollisions(float3 *positions, float3 **linearMomenta, const int &numberOfPrimitives, const int &numberOfThreads)
{
	unsigned int *mortonCodes;
	hipError_t cudaStatus = hipMalloc((void**)&mortonCodes, numberOfPrimitives * sizeof(unsigned int));
	//cudaStatus = hipMemset(mortonCodes, 0, sizeof(unsigned int) * numberOfPrimitives);
	if (cudaStatus != hipSuccess) {
		hipFree(mortonCodes);
		return cudaFail(cudaStatus, "cudaMalloc_mortonCodes");
	}

	//assign a Morton code to each primitive
	//launch all objects
	generateMortonCodes << <(numberOfPrimitives + numberOfThreads - 1) / numberOfThreads, numberOfThreads >> >(positions, mortonCodes, numberOfPrimitives);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		hipFree(mortonCodes);
		return cudaFail(cudaStatus, "generateMortonCodes_getLastError");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		hipFree(mortonCodes);
		return cudaFail(cudaStatus, "generateMortonCodes_cudaDeviceSynchronize");
	}

	//create leaf nodes here
	//then sort them using their morton codes as keys
	//and pass them as argument to the BVH hierarchy creation routine
	Primitive *leafNodes;
	cudaStatus = hipMalloc((void**)&leafNodes, numberOfPrimitives * sizeof(Primitive));
	//cudaStatus = hipMemset(leafNodes, 0, sizeof(Primitive) * numberOfPrimitives);
	if (cudaStatus != hipSuccess){
		hipFree(mortonCodes);
		hipFree(leafNodes);
		return cudaFail(cudaStatus, "cudaMalloc_leafNodes");
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess){
		hipFree(mortonCodes);
		hipFree(leafNodes);
		return cudaFail(cudaStatus, "testingMemoryAccess_cudaDeviceSynchronize");
	}



	constructLeafNodes << <(numberOfPrimitives + numberOfThreads - 1) / numberOfThreads, numberOfThreads >> >(leafNodes, positions, (float*)(*linearMomenta), numberOfPrimitives);	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		hipFree(mortonCodes);
		hipFree(leafNodes);
		return cudaFail(cudaStatus, "constructLeafNodes_cudaGetLastError");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess){
		hipFree(mortonCodes);
		hipFree(leafNodes);
		return cudaFail(cudaStatus, "constructLeafNodes_cudaDeviceSynchronize");
	}

	//exit(1);
	//sorting procedure using cub (currently building)
	hipcub::DoubleBuffer<unsigned int> sortKeys; //keys to sort by - Morton codes
	hipcub::DoubleBuffer<Primitive> sortValues; //also sort corresponding particles by key
	
	//presumambly, there is no need to allocate space for the current buffers
	sortKeys.d_buffers[0] = mortonCodes;
	sortValues.d_buffers[0] = leafNodes;

	hipcub::CachingDeviceAllocator  g_allocator(true);

	cudaStatus = g_allocator.DeviceAllocate((void**)&sortKeys.d_buffers[1], sizeof(unsigned int) * numberOfPrimitives);
	if (cudaStatus != hipSuccess)
	{
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		return cudaFail(cudaStatus, "sortKeys_gAllocate");
	}
	
	cudaStatus = g_allocator.DeviceAllocate((void**)&sortValues.d_buffers[1], sizeof(Primitive) * numberOfPrimitives);
	if (cudaStatus != hipSuccess)
	{
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "sortValues_gAllocate");
	}

	// Allocate temporary storage
	size_t  temp_storage_bytes = 0;
	void    *d_temp_storage = NULL;
	cudaStatus = hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, sortKeys, sortValues, numberOfPrimitives);
	if (cudaStatus != hipSuccess)
	{
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "first call to DeviceRadixSort");
	}
	cudaStatus = g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
	if (cudaStatus != hipSuccess)
	{
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "cub: temporary storage alocation");
	}

	// Run sort
	//Note: why do I need to sort the particles themselves?
	//The code I found does nothing of the kind.
	cudaStatus = hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, sortKeys, sortValues, numberOfPrimitives);
	if (cudaStatus != hipSuccess)
	{
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "second call to DeviceRadixSort");
	}
	
	//sort seems to be working properly

	Primitive* internalNodes;	
	cudaStatus = hipMalloc((void**)&internalNodes, (numberOfPrimitives - 1) * sizeof(Primitive));
	//cudaStatus = hipMemset(internalNodes, 0, sizeof(Primitive) * numberOfPrimitives);
	if (cudaStatus != hipSuccess) {
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "cudaMalloc_internalNodes");
	}
	cudaStatus = generateHierarchy(internalNodes, sortValues.Current(), sortKeys.Current(), numberOfPrimitives);
	if (cudaStatus != hipSuccess){
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "bvh_generateHierarchy");
	}

	handleCollisions << <(numberOfPrimitives + numberOfThreads - 1) / numberOfThreads, numberOfThreads >> >(leafNodes, numberOfPrimitives);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "constructLeafNodes_cudaGetLastError");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess){
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "constructLeafNodes_cudaDeviceSynchronize");
	}

	cudaStatus = update(leafNodes, 0.1, numberOfPrimitives, numberOfThreads);
	if (cudaStatus != hipSuccess){
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "update_cudaGetLastError");
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess){
		hipFree(internalNodes);
		hipFree(d_temp_storage);
		hipFree(mortonCodes);
		hipFree(leafNodes);
		g_allocator.DeviceFree(sortKeys.d_buffers[1]);
		g_allocator.DeviceFree(sortValues.d_buffers[1]);
		return cudaFail(cudaStatus, "update_cudaDeviceSynchronize");
	}
	//cleaning up
	hipFree(internalNodes);
	hipFree(d_temp_storage);
	hipFree(mortonCodes);
	hipFree(leafNodes);
	g_allocator.DeviceFree(sortKeys.d_buffers[1]);
	g_allocator.DeviceFree(sortValues.d_buffers[1]);
	//g_allocator.DeviceFree(sortKeys.d_buffers[0]);
	//g_allocator.DeviceFree(sortValues.d_buffers[0]);
	return hipSuccess;
}

hipError_t cudaFail(hipError_t cudaStatus, char *funcName)
{
	std::cout << "Callback function: " << funcName << std::endl;
	std::cout << "Error code: " << cudaStatus << std::endl;
	std::cout << "Error type: " << hipGetErrorString(cudaStatus) << std::endl;
	return cudaStatus;
}
