#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include "hip/hip_runtime.h"
#include ""
#include <cuda_gl_interop.h>
/*#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"*/
#include <stdio.h>

hipError_t animateWithCuda(struct hipGraphicsResource* testingVBO_CUDA, size_t *num_bytes, double offset);

__global__ void animateKernel(float3* positions, double offset)
{
	int i = threadIdx.y * blockDim.x + threadIdx.x;
	int j = threadIdx.y;
	positions[i].x += offset;
	positions[i].y = (float)j;
	positions[i].z = -10.0;
}

// Helper function for using CUDA to animate particles.
hipError_t animateWithCuda(struct hipGraphicsResource* testingVBO_CUDA, size_t *num_bytes, double offset)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	/*cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}*/
	float3* positions;

	cudaStatus = hipGraphicsMapResources(1, &testingVBO_CUDA, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsMapResources returned error code %d before launching animateKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGraphicsResourceGetMappedPointer((void**)&positions, num_bytes, testingVBO_CUDA);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsResourceGetMappedPointer returned error code %d before launching animateKernel!\n", cudaStatus);
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	animateKernel << <2, 512 >> >(positions, offset);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initializeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching animateKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGraphicsUnmapResources(1, &testingVBO_CUDA, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGraphicsUnmapResources returned error code %d after launching animateKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	return cudaStatus;
}
