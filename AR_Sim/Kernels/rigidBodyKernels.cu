#include "hip/hip_runtime.h"
/*
* Particle-based dynamic simulations for rigid bodies.
* Instead of having autonomous particles, we now associate each of them with a rigid body.
* For each particle we have a relative position to the center of mass of its associated rigid body.
* System integration occures over each rigid body.
*/
#include "BVHAuxiliary.cuh"
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#ifndef GLM_FORCE_CUDA
#define GLM_FORCE_CUDA
#endif
#include <glm/glm.hpp>
#include <glm/gtc/quaternion.hpp>

#define HIPCUB_STDERR
//define _HipcubLog to avoid encountering error: "undefined reference"
#if !defined(_HipcubLog)
#if (HIPCUB_ARCH == 0)
#define _HipcubLog(format, ...) printf(format,__VA_ARGS__);
#elif (HIPCUB_ARCH >= 200)
#define _HipcubLog(format, ...) printf("[block (%d,%d,%d), thread (%d,%d,%d)]: " format, blockIdx.z, blockIdx.y, blockIdx.x, threadIdx.z, threadIdx.y, threadIdx.x, __VA_ARGS__);
#endif
#endif
#define inf 0x7f800000
//cub headers
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_reduce.cuh>

/*
* Integrates each rigid body. Moves center of mass only.
*/
__global__ void integrateRigidBody(float4 *CMs, //rigid body center of mass
	float4 *vel, //velocity of rigid body
	float4 *force, //force applied to rigid body due to previous collisions
	float4 *rbAngularVelocity, //contains angular velocities for each rigid body
	glm::quat *rbQuaternion, //contains current quaternion for each rigid body
	float4 *rbTorque, //torque applied to rigid body due to previous collisions
	float4 *rbAngularMomentum, //cumulative angular momentum of the rigid body
	float4 *rbLinearMomentum, //cumulative linear momentum of the rigid body
	glm::mat3 *rbInertia, //original moment of inertia for each rigid body - 9 values per RB
	glm::mat3 *rbCurrentInertia, //current moment of inertia for each rigid body - 9 values per RB
	glm::vec3 *rbAngularAcceleration, //current angular acceleration due to misaligned angular momentum and velocity
	float deltaTime, //dt
	float *rbRadii, //radius chosen for each rigid body sphere
	float *rbMass, //inverse of total mass of rigid body
	float3 minPos, //smallest coordinate of scene's bounding box
	float3 maxPos, //largest coordinate of scene's bounding box
	int numBodies, //number of rigid bodies
	SimParams params) //simulation parameters
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= numBodies)
		return;
	float4 locLinearMomentum = rbLinearMomentum[index];
	locLinearMomentum += force[index] * deltaTime;
	locLinearMomentum += make_float4(params.gravity, 0) * deltaTime;

	maxPos.x = maxPos.x + 0.1;
	maxPos.y = maxPos.y + 0.1;
	maxPos.z = maxPos.z + 0.1;

	minPos.x = minPos.x - 0.1;
	minPos.y = minPos.y - 0.1;
	minPos.z = minPos.z - 1;

	float4 locPos = CMs[index];
	float locMass = rbMass[index];
	float sphereRadius = rbRadii[index];
	if (locPos.x > maxPos.x - sphereRadius)
	{
		locPos.x = maxPos.x - sphereRadius;
		locLinearMomentum.x *= params.boundaryDamping;
	}

	if (locPos.x < minPos.x + sphereRadius)
	{
		locPos.x = minPos.x + sphereRadius;
		locLinearMomentum.x *= params.boundaryDamping;
	}

	if (locPos.y > maxPos.y - sphereRadius && locLinearMomentum.y > 0)
	{
		locPos.y = maxPos.y - 2 * sphereRadius;
		locLinearMomentum.y *= params.boundaryDamping;
	}

	if (locPos.y < minPos.y + sphereRadius)
	{
		locPos.y = minPos.y + sphereRadius;
		locLinearMomentum.y *= params.boundaryDamping;
	}

	if (locPos.z > maxPos.z - sphereRadius)
	{
		locPos.z = maxPos.z - sphereRadius;
		locLinearMomentum.z *= params.boundaryDamping;
	}

	if (locPos.z < minPos.z + sphereRadius)
	{
		locPos.z = minPos.z + sphereRadius;
		locLinearMomentum.z *= params.boundaryDamping;
	}

	locLinearMomentum *= params.globalDamping;
	float4 locVel = locLinearMomentum * locMass;
	rbLinearMomentum[index] = locLinearMomentum;
	//locVel += make_float4(params.gravity, 0) * locMass * deltaTime;
	//locVel *= params.globalDamping;

	locPos += locVel * deltaTime;

	//add a 1cm offset to prevent false collisions
	locPos.w = 0.f;
	locVel.w = 0.f;
	CMs[index] = locPos;
	vel[index] = locVel;
	force[index] = make_float4(0, 0, 0, 0); //reset force to zero
	
	//now consider rotational motion
	glm::mat3 inertia = rbInertia[index]; //each inertia matrix has 9 elements

	glm::quat quaternion = rbQuaternion[index];
	glm::mat3 rot = mat3_cast(quaternion);

	glm::mat3 currentInertia = rot * inertia * transpose(rot);
	float4 angularMomentum = rbAngularMomentum[index];
	float4 torque = rbTorque[index];
	angularMomentum += torque * deltaTime;
	//angularMomentum *= params.globalDamping;
	glm::vec3 currentMomentum = glm::vec3(angularMomentum.x, angularMomentum.y, angularMomentum.z);
	glm::vec3 newVelocity = currentInertia * currentMomentum;
	//	correct angular drift
	glm::vec3 currentTorque(torque.x, torque.y, torque.z);
	glm::vec3 angularAcceleration = currentInertia * glm::cross(currentMomentum, newVelocity);

//	newVelocity -= angularAcceleration * deltaTime;
//	newVelocity = glm::vec3(0.001, 0.004, 0.001);
	glm::quat qdot = glm::quat(0, newVelocity.x, newVelocity.y, newVelocity.z) * quaternion;
	qdot /= 2.f;
	quaternion += qdot * deltaTime;
//	float angular_speed = glm::length(newVelocity);
//	float rotation_angle = angular_speed*deltaTime;
//	glm::vec3 rotationAxis = normalize(newVelocity);
//	glm::quat dq(cos(rotation_angle / 2), sin(rotation_angle / 2) * rotationAxis.x, sin(rotation_angle / 2) * rotationAxis.y, sin(rotation_angle / 2) * rotationAxis.z);
//	quaternion = glm::cross(dq, quaternion);
	quaternion = normalize(quaternion);

	newVelocity -= angularAcceleration * deltaTime;

	rbAngularAcceleration[index] = angularAcceleration;
	rbCurrentInertia[index] = currentInertia;
	rbAngularMomentum[index] = angularMomentum;
	rbQuaternion[index] = quaternion;
	rbAngularVelocity[index] = make_float4(newVelocity.x, newVelocity.y, newVelocity.z, 0);
	rbTorque[index] = make_float4(0, 0, 0, 0); //reset torque to zero
}

void integrateSystemRigidBodies(float4 *CMs, //rigid body center of mass
	float4 *vel, //velocity of rigid body
	float4 *force, //force applied to rigid body due to previous collisions
	float4 *rbAngularVelocity, //contains angular velocities for each rigid body
	glm::quat *rbQuaternion, //contains current quaternion for each rigid body
	float4 *rbTorque, //torque applied to rigid body due to previous collisions
	float4 *rbAngularMomentum, //cumulative angular momentum of the rigid body
	float4 *rbLinearMomentum, //cumulative linear momentum of the rigid body
	glm::mat3 *rbInertia, //original moment of inertia for each rigid body - 9 values per RB
	glm::mat3 *rbCurrentInertia, //current moment of inertia for each rigid body - 9 values per RB
	glm::vec3 *rbAngularAcceleration, //current angular acceleration due to misaligned angular momentum and velocity
	float deltaTime, //dt
	float *rbRadii, //radius chosen for each rigid body sphere
	float *rbMass, //total mass of rigid body
	float3 minPos, //smallest coordinate of scene's bounding box
	float3 maxPos, //largest coordinate of scene's bounding box
	int numBodies, //number of rigid bodies
	SimParams params, //simulation parameters
	int numThreads) //number of threads
{
	dim3 blockDim(numThreads, 1);
	dim3 gridDim((numBodies + numThreads - 1) / numThreads, 1);
	if (gridDim.x < 1)
		gridDim.x = 1;
	integrateRigidBody << < gridDim, blockDim >> >(CMs, //rigid body center of mass
		vel, //velocity of rigid body
		force, //force applied to rigid body due to previous collisions
		rbAngularVelocity, //contains angular velocities for each rigid body
		rbQuaternion, //contains current quaternion for each rigid body
		rbTorque, //torque applied to rigid body due to previous collisions
		rbAngularMomentum, //cumulative angular momentum of the rigid body
		rbLinearMomentum, //cumulative linear momentum of the rigid body
		rbInertia, //original moment of inertia for each rigid body - 9 values per RB
		rbCurrentInertia, //current moment of inertia for each rigid body - 9 values per RB
		rbAngularAcceleration, //current angular acceleration due to misaligned angular momentum and velocity
		deltaTime, //dt
		rbRadii, //radius chosen for each rigid body sphere
		rbMass, //total mass of rigid body
		minPos, //smallest coordinate of scene's bounding box
		maxPos, //largest coordinate of scene's bounding box
		numBodies, //number of rigid bodies
		params); //simulation parameters
}

/*
* Function to calculate the global position of each particle given its relative position to its
* associated rigid body's center of mass.
*/
__global__ void computeGlobalAttributes(float4 *CMs, //rigid body's center of mass
	float4 *rigidVel, //rigid body's velocity
	float4 *relativePos, //particle's relative position
	float4 *globalPos, //particle's global position
	float4 *globalVel, //particle's world velocity
	glm::quat *rbQuaternion, //contains current quaternion for each rigid body
	float4 *rbAngularVelocity, //contains angular velocities for each rigid body
	int *rigidBodyIndex, //index of associated rigid body
	int numParticles) //number of particles
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;
	int correspondingRigidBody = rigidBodyIndex[index];
	if (correspondingRigidBody == -1) return; //if this is an independent virtual particle
	//float4 tempQuat = rbQuaternion[correspondingRigidBody];
	//glm::quat quaternion(tempQuat.w, tempQuat.x, tempQuat.y, tempQuat.z);
	glm::quat quaternion = rbQuaternion[correspondingRigidBody];
	float4 tempPos = relativePos[index];
	glm::vec4 pos = glm::vec4(tempPos.x, tempPos.y, tempPos.z, tempPos.w);
	glm::mat4 rot = mat4_cast(quaternion);
	pos = rot * pos;
	//pos = quaternion * pos * conjugate(quaternion);
	tempPos = make_float4(pos.x, pos.y, pos.z, pos.w);
	relativePos[index] = tempPos;
	globalPos[index] = tempPos + CMs[correspondingRigidBody];
	//particle's velocity is the same as its associated rigid body's
	//for the moment we ignore angular velocity
	globalVel[index] = rigidVel[correspondingRigidBody] + make_float4(cross(make_float3(rbAngularVelocity[correspondingRigidBody]), make_float3(tempPos)), 0);
}

void computeGlobalAttributesWrapper(float4 *CMs, //rigid body's center of mass
float4 *rigidVel, //rigid body's velocity
float4 *relativePos, //particle's relative position
float4 *globalPos, //particle's global position
float4 *globalVel, //particle's world velocity
glm::quat *rbQuaternion, //contains current quaternion for each rigid body
float4 *rbAngularVelocity, //contains angular velocities for each rigid body
int *rigidBodyIndex, //index of associated rigid body
int numParticles, //number of particles
int numThreads) //number of threads
{
	dim3 blockDim(numThreads, 1);
	dim3 gridDim((numParticles + numThreads - 1) / numThreads, 1);
	computeGlobalAttributes << < gridDim, blockDim >> >(CMs, //rigid body's center of mass
		rigidVel, //rigid body's velocity
		relativePos, //particle's relative position
		globalPos, //particle's global position
		globalVel, //particle's world velocity
		rbQuaternion, //contains current quaternion for each rigid body
		rbAngularVelocity, //contains angular velocities for each rigid body
		rigidBodyIndex, //index of associated rigid body
		numParticles); //number of particles
}


/*
 * Kernel function used to pre-load necessary rigid body variables used by different particles during
 * collision detection (see wrapper function for more details).
 */
__global__ void PreloadRigidBodyVariablesKernel(
		float4 *rbForce, //Input: rigid body forces - one element per rigid body
		float4 *rbTorque, //Input: rigid body torques - one element per rigid body
		float4 *rbPositions, //Input: rigid body center of mass - one element per rigid body
		float4 *pForce, //Output: rigid body forces - one element per particle
		float4 *pTorque, //Output: rigid body torques - one element per particle
		float4 *pPositions, //Output: rigid body center of mass - one element per particle
		int *rbIndices, //Auxil.: indices of corresponding rigid bodies - one element per particle
		int numParticles) //Auxil.: number of particles)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;

	int rigidBodyIndex = rbIndices[index];
	if (rigidBodyIndex == - 1) //if this is an independent particle
		return; //no values are stored - check explicitly @ collision detection

	pForce[index] = rbForce[rigidBodyIndex]; //load rigid body force
	pTorque[index] = rbTorque[rigidBodyIndex]; //load rigid body torque
	pPositions[index] = rbPositions[rigidBodyIndex]; //load rigid body positions

}

/*
 * Since some rigid body variables are both accessed and, most importantly, updated by all of their respective
 * particles, it is necessary to synchronize these activities. Since it is impossible to specifically match ea-
 * ch block to a particular rigid body, so as to utilize shared memory, we must explicitly pre-load these vari-
 * ables. This is the purpose of this function. Its arguments are /float4 rbForce, /float4 rbTorque and /float4
 * rbPositions. This list must be updated if more variables are added. It has an equal number of outputs, but
 * each output array has a size equal to the total number of particles. Specific care must be given to the bogus
 * rigid body values corresponding to independent virtual particles. They should be explicitly checked during col-
 * lision detectin. After processing collisions, the results must be gathered (i.e. reduced) to these variables again.
 * Possibly DEPRECATED.
 */
void PreloadRigidBodyVariablesWrapper(
		float4 *rbForce, //Input: rigid body forces - one element per rigid body
		float4 *rbTorque, //Input: rigid body torques - one element per rigid body
		float4 *rbPositions, //Input: rigid body positions - one element per rigid body
		float4 *pForce, //Output: rigid body forces - one element per particle
		float4 *pTorque, //Output: rigid body torques - one element per particle
		float4 *pPositions, //Output: rigid body positions - one element per particle
		int *rbIndices, //Auxil.: indices of corresponding rigid bodies - one element per particle
		int numParticles, //Auxil.: number of particles
		int numThreads) //number of threads to use
{
	dim3 blockDim(numThreads, 1);
	dim3 gridDim((numParticles + numThreads - 1) / numThreads, 1);
	PreloadRigidBodyVariablesKernel << < gridDim, blockDim >> >(
			rbForce, //Input: rigid body forces - one element per rigid body
			rbTorque, //Input: rigid body torques - one element per rigid body
			rbPositions, //Input: rigid body positions - one element per rigid body
			pForce, //Output: rigid body forces - one element per particle
			pTorque, //Output: rigid body torques - one element per particle
			pPositions, //Output: rigid body positions - one element per particle
			rbIndices, //Auxil.: indices of corresponding rigid bodies - one element per particle
			numParticles); //Auxil.: number of particles)
}


struct CustomAdd
{
	template <typename T>
	__device__ __forceinline__
	T operator()(const T &a, const T &b) { return a + b; }
};

/*
 * Kernel used to reset particle force, torque and correct position values for current iteration to zero.
 */
__global__ void resetBlockVariables(
		float4 *pForce, //Input: rigid body forces - one element per particle
		float4 *pTorque, //Input: rigid body torques - one element per particle
		float4 *pPositions, //Input: rigid body positions - one element per particle
		int numParticles)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;
	pForce[index] = make_float4(0.f);
	pTorque[index] = make_float4(0.f);
	pPositions[index] = make_float4(0.f);
}

/*
 * Kernel used to add reduced results to rigid body variables.
 */
__global__ void combineIntermediateResults(
		float4 *rbForce, //Output: rigid body forces - one element per rigid body
		float4 *rbTorque, //Output: rigid body torques - one element per rigid body
		float4 *rbPositions, //Output: rigid body positions - one element per rigid body
		float4 *intermediateForce, //Input: reduced rigid body forces - one element per rigid body
		float4 *intermediateTorque, //Input: reduced rigid body torques - one element per rigid body
		float4 *intermediatePositions, //Input: reduced rigid body positions - one element per rigid body
		int numRigidBodies) //Auxil.: number of rigid bodies
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodies)
		return;

	//simply add corresponding intermediate results
	rbForce[index] += intermediateForce[index];
	rbTorque[index] += intermediateTorque[index];
	//rbPositions[index] += intermediatePositions[index];
}

struct CustomMax4RB
{
	template <typename T>
	__device__ __forceinline__
	T operator()(const T &a, const T &b) const {
		T res;
		res.x =  (b.x > a.x) ? b.x : a.x;
		res.y =  (b.y > a.y) ? b.y : a.y;
		res.z =  (b.z > a.z) ? b.z : a.z;
		res.w =  (b.w > a.w) ? b.w : a.w;
		return res;
	}
};

/*
 * After computing collision detection results for each particles it is necessary to combine (i.e. reduce)
 * all the intermediate results for each rigid body. This is done using cub for reduction. To do this it is
 * necessary to know how many particles belong to each rigid body, and the start and end of each rigid body's
 * particle sequence. To this end, we introduce two new variables, /int* particlesPerObjectThrown and /int objectsThrown
 * to keep count of how many objects are thrown, and how many particles belong to each of them. Note: objectsThrown is
 * different than numRigidBodies because it also takes into account point sprites.
 */
void ReduceRigidBodyVariables(
		float4 *rbForce, //Output: rigid body forces - one element per rigid body
		float4 *rbTorque, //Output: rigid body torques - one element per rigid body
		float4 *rbPositions, //Output: rigid body positions - one element per rigid body
		float4 *pForce, //Input: rigid body forces - one element per particle
		float4 *pTorque, //Input: rigid body torques - one element per particle
		float4 *pPositions, //Input: rigid body positions - one element per particle
		int *particlesPerObjectThrown, //Auxil.: number of particles for each rigid body - one element per thrown objects
		bool *isRigidBody, //Auxil.: flag indicating whether thrown object is a rigid body
		int objectsThrown, //Auxil.: number of objects thrown - rigid bodies AND point sprites
		int numRigidBodies, //Auxil.: number of rigid bodies
		int numThreads, //number of threads to use
		int numParticles, //total number of virtual particles
		bool *toExit)
{
	//auxiliary variables used to store intermediate reduce results
	float4 *intermediateForce;
	float4 *intermediateTorque;
	float4 *intermediatePositions;

	hipMalloc((void**)&intermediateForce, sizeof(float4) * numRigidBodies);
	hipMalloc((void**)&intermediateTorque, sizeof(float4) * numRigidBodies);
	hipMalloc((void**)&intermediatePositions, sizeof(float4) * numRigidBodies);
//	std::cout << "Started new iteration..." << std::endl;
//	std::cout << "Number of objects thrown: " << objectsThrown << std::endl;
//	std::cout << "Number of rigid bodies: " << numRigidBodies << std::endl;
//	std::cout << "Number of particles: " << numParticles << std::endl;
	int rbCounter = 0;
	int sumParticles = 0;
	//bool toExit = false;
	for (int num = 0; num < objectsThrown; num++)
	{
		int currentNumParticles = particlesPerObjectThrown[num];
		if(isRigidBody[num]) //if this is NOT a point sprite - point sprites are not rigid bodies
		{
//			std::cout << "Current rigid body: " << rbCounter + 1 << std::endl;
//			std::cout << "Particles of current rigid body: " << currentNumParticles << std::endl;
//			std::cout << "Total particles processed so far: " << sumParticles << std::endl;
//			float4 *cpuForce = new float4[currentNumParticles];
//			checkCudaErrors(hipMemcpy(cpuForce, &pForce[sumParticles], sizeof(float4) * currentNumParticles, hipMemcpyDeviceToHost));
//			for (int i = 0; i < currentNumParticles; i++)
//				if (cpuForce[i].x != 0 || cpuForce[i].y != 0 ||cpuForce[i].z != 0)
//					std::cout << "Particle force @" << i + sumParticles << ": (" << cpuForce[i].x << " " << cpuForce[i].y << " " << cpuForce[i].z << ")" << std::endl;
//			delete cpuForce;
//			CustomAdd addOp;
//			float4 init = make_float4(0, 0, 0, 0);
			float4 *d_out;
			//rigid body force reduction
			checkCudaErrors(hipMalloc((void**)&d_out, sizeof(float4) * currentNumParticles));
			void     *d_temp_storage = NULL;
			size_t   temp_storage_bytes = 0;
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pForce[sumParticles], d_out, currentNumParticles));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pForce[sumParticles], d_out, currentNumParticles));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediateForce[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));
			float4 cpuTest;
			checkCudaErrors(hipMemcpy(&cpuTest, &intermediateForce[rbCounter], sizeof(float4), hipMemcpyDeviceToHost));
			if (cpuTest.x != 0 || cpuTest.y != 0 ||cpuTest.z != 0)
			{
//				std::cerr << "Reduced force: (" << cpuTest.x << " " << cpuTest.y << " " << cpuTest.z << ")" << std::endl;
				if (cpuTest.x != cpuTest.x || cpuTest.y != cpuTest.y || cpuTest.z != cpuTest.z)
					*toExit = true;
			}
			hipMemset(d_out, 0, particlesPerObjectThrown[num]);

			//rigid body torque reduction
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pTorque[sumParticles], d_out, currentNumParticles));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pTorque[sumParticles], d_out, currentNumParticles));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediateTorque[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));
			checkCudaErrors(hipMemcpy(&cpuTest, &intermediateTorque[rbCounter], sizeof(float4), hipMemcpyDeviceToHost));
			if (cpuTest.x != 0 || cpuTest.y != 0 ||cpuTest.z != 0)
			{
//				std::cerr << "Applied torque: (" << cpuTest.x << " " << cpuTest.y << " " << cpuTest.z << ")" << std::endl;
				if (cpuTest.x != cpuTest.x || cpuTest.y != cpuTest.y || cpuTest.z != cpuTest.z)
					*toExit = true;
			}
			hipMemset(d_out, 0, particlesPerObjectThrown[num]);

			CustomMax4RB    max_op;
			float4 init = make_float4(-inf, -inf, -inf, -inf);

			//rigid body positions reduction
			checkCudaErrors(hipcub::DeviceReduce::Reduce(d_temp_storage,
					temp_storage_bytes,
					&pPositions[sumParticles],
					d_out,
					currentNumParticles,
					max_op,
					init));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Reduce(d_temp_storage,
					temp_storage_bytes,
					&pPositions[sumParticles],
					d_out,
					currentNumParticles,
					max_op,
					init));


			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediatePositions[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));
//			checkCudaErrors(hipMemcpy(&cpuTest, &intermediatePositions[rbCounter], sizeof(float4), hipMemcpyDeviceToHost));
//			std::cout << "Applied correction: (" << cpuTest.x << " " << cpuTest.y << " " << cpuTest.z << ")" << std::endl;
			checkCudaErrors(hipFree(d_out));
			checkCudaErrors(hipFree(d_temp_storage));
			rbCounter++; //increase counter to keep track of rigid bodies processed


		}
		sumParticles += currentNumParticles;
	}

	dim3 blockDim(numThreads, 1);
	dim3 gridDim((numRigidBodies + numThreads - 1) / numThreads, 1);
	if (gridDim.x < 1)
			gridDim.x = 1;
	if(numRigidBodies)
	{

		combineIntermediateResults<< < gridDim, blockDim >> >(
				rbForce, //Output: rigid body forces - one element per rigid body
				rbTorque, //Output: rigid body torques - one element per rigid body
				rbPositions, //Output: rigid body positions - one element per rigid body
				intermediateForce, //Input: reduced rigid body forces - one element per rigid body
				intermediateTorque, //Input: reduced rigid body torques - one element per rigid body
				intermediatePositions, //Input: reduced rigid body positions - one element per rigid body
				numRigidBodies); //Auxil.: number of rigid bodies
//		float *totalTorque = new float[4 * numRigidBodies];
//		float *currentTorque = new float[4 * numRigidBodies];
//		hipMemcpy(totalTorque, rbTorque, sizeof(float) * 4 *numRigidBodies, hipMemcpyDeviceToHost);
//		hipMemcpy(currentTorque, intermediateTorque, sizeof(float) * 4 *numRigidBodies, hipMemcpyDeviceToHost);
//		for (int i = 0; i < numRigidBodies; i++)
//		{
//			bool closeAll = false;
//			if(totalTorque[4 * i] != totalTorque[4 * i] ||
//					totalTorque[4 * i + 1] != totalTorque[4 * i + 1] ||
//					totalTorque[4 * i + 2] != totalTorque[4 * i + 2])
//			{
//				std::cerr << "Total torque is wrong." << std::endl;
//				closeAll = true;
//			}
//			if(currentTorque[4 * i] != currentTorque[4 * i] ||
//					currentTorque[4 * i + 1] != currentTorque[4 * i + 1] ||
//					currentTorque[4 * i + 2] != currentTorque[4 * i + 2])
//			{
//				std::cerr << "Current torque is wrong." << std::endl;
//				closeAll = true;
//			}
//
//			if (closeAll)
//			{
//				exit(1);
//			}
//		}
//		delete totalTorque;
//		delete currentTorque;
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	blockDim = dim3(numThreads, 1);
	gridDim = dim3((numParticles + numThreads - 1) / numThreads, 1);
	if (gridDim.x < 1)
		gridDim.x = 1;
	if(numParticles)
	{
		resetBlockVariables<< < gridDim, blockDim >> >(
				pForce, //Input: rigid body forces - one element per particle
				pTorque, //Input: rigid body torques - one element per particle
				pPositions, //Input: rigid body positions - one element per particle
				numParticles);
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	hipFree(intermediateForce);
	hipFree(intermediateTorque);
	hipFree(intermediatePositions);
}

void testCubReduce(int elements)
{
	float4 *cpuData = new float4[elements];
	for (int i = 0; i < elements; i++)
	{
		cpuData[i].x = 1.0;
		cpuData[i].y = 2.0;
		cpuData[i].z = 3.0;
		cpuData[i].w = -1.0;
	}
	float4 *gpuData;
	hipMalloc((void**)&gpuData, sizeof(float4) * elements);
	hipMemcpy(gpuData, cpuData, sizeof(float4) * elements, hipMemcpyHostToDevice);

	float4 *d_out;
	checkCudaErrors(hipMalloc((void**)&d_out, sizeof(float4) * elements));
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;
	checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpuData, d_out, elements));
	checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
	checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpuData, d_out, elements));

	float4 gpuResult;
	//copy result CPU
	checkCudaErrors(hipMemcpy(&gpuResult, &d_out[0], sizeof(float4), hipMemcpyDeviceToHost));
	std::cout << "Result of cub sum reduction is: (" << gpuResult.x << ", " << gpuResult.y << ", " << gpuResult.z <<
			", " << gpuResult.w << ")" << std::endl;

	hipFree(d_out);
	hipFree(d_temp_storage);
	hipFree(gpuData);
	delete cpuData;
}



/*
 * Kernel used to reset particle force, torque and correct position values for current iteration to zero.
 */
__global__ void resetBlockVariablesAR(
		float4 *pForce, //Input: rigid body forces - one element per particle
		float4 *pTorque, //Input: rigid body torques - one element per particle
		float4 *pPositions, //Input: rigid body positions - one element per particle
		int *pCountARCollions, //Input: AR collisions - one element per particle
		int numParticles)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numParticles)
		return;
	pForce[index] = make_float4(0.f);
	pTorque[index] = make_float4(0.f);
	pPositions[index] = make_float4(0.f);
	pCountARCollions[index] = 0;
}

/*
 * Kernel used to add reduced results to rigid body variables.
 */
__global__ void combineIntermediateResultsAR(
		float4 *rbForce, //Output: rigid body forces - one element per rigid body
		float4 *rbTorque, //Output: rigid body torques - one element per rigid body
		float4 *rbPositions, //Output: rigid body positions - one element per rigid body
		float4 *intermediateForce, //Input: reduced rigid body forces - one element per rigid body
		float4 *intermediateTorque, //Input: reduced rigid body torques - one element per rigid body
		float4 *intermediatePositions, //Input: reduced rigid body positions - one element per rigid body
		int *ARCollisionsRigidBody, //Input: reduced number of collisions
		int numRigidBodies) //Auxil.: number of rigid bodies
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodies)
		return;
	float numCollisions = (float)ARCollisionsRigidBody[index];
	if (numCollisions < 1) numCollisions = 1;
	//simply add corresponding intermediate results
	rbForce[index] += intermediateForce[index] / numCollisions;
	rbTorque[index] += intermediateTorque[index] / numCollisions;// * 4.f;
	//rbPositions[index] += intermediatePositions[index];
}
/*
 * This function is different to the original in that it now averages computed force/torque over
 * the number of collisions per rigid body to normalize the total force/torque.
 * TODO: integrate the common parts of these two function in one auxiliary function to avoid
 * unnecessary debugging.
 */
void ReduceRigidBodyARVariables(
		float4 *rbForce, //Output: rigid body forces - one element per rigid body
		float4 *rbTorque, //Output: rigid body torques - one element per rigid body
		float4 *rbPositions, //Output: rigid body positions - one element per rigid body
		float4 *pForce, //Input: rigid body forces - one element per particle
		float4 *pTorque, //Input: rigid body torques - one element per particle
		float4 *pPositions, //Input: rigid body positions - one element per particle
		int *pCountARCollions, //Input: AR collisions - one element per particle
		int *particlesPerObjectThrown, //Auxil.: number of particles for each rigid body - one element per thrown objects
		bool *isRigidBody, //Auxil.: flag indicating whether thrown object is a rigid body
		int objectsThrown, //Auxil.: number of objects thrown - rigid bodies AND point sprites
		int numRigidBodies, //Auxil.: number of rigid bodies
		int numThreads, //number of threads to use
		int numParticles, //total number of virtual particles
		bool *toExit)
{
	//auxiliary variables used to store intermediate reduce results
	float4 *intermediateForce;
	float4 *intermediateTorque;
	float4 *intermediatePositions;

	hipMalloc((void**)&intermediateForce, sizeof(float4) * numRigidBodies);
	hipMalloc((void**)&intermediateTorque, sizeof(float4) * numRigidBodies);
	hipMalloc((void**)&intermediatePositions, sizeof(float4) * numRigidBodies);

	int *ARCollisionsRigidBody;
	hipMalloc((void**)&ARCollisionsRigidBody, sizeof(int) * numRigidBodies);

	int rbCounter = 0;
	int sumParticles = 0;
	for (int num = 0; num < objectsThrown; num++)
	{
		int currentNumParticles = particlesPerObjectThrown[num];
		if(isRigidBody[num]) //if this is NOT a point sprite - point sprites are not rigid bodies
		{
			float4 *d_out;
			checkCudaErrors(hipMalloc((void**)&d_out, sizeof(float4) * currentNumParticles));
			void     *d_temp_storage = NULL;
			size_t   temp_storage_bytes = 0;
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pForce[sumParticles], d_out, currentNumParticles));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pForce[sumParticles], d_out, currentNumParticles));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediateForce[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));
			float4 cpuTest;
			checkCudaErrors(hipMemcpy(&cpuTest, &intermediateForce[rbCounter], sizeof(float4), hipMemcpyDeviceToHost));
			if (cpuTest.x != 0 || cpuTest.y != 0 ||cpuTest.z != 0)
			{
//				std::cerr << "Reduced force: (" << cpuTest.x << " " << cpuTest.y << " " << cpuTest.z << ")" << std::endl;
				if (cpuTest.x != cpuTest.x || cpuTest.y != cpuTest.y || cpuTest.z != cpuTest.z)
					*toExit = true;
			}
			hipMemset(d_out, 0, particlesPerObjectThrown[num]);

			//rigid body torque reduction
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pTorque[sumParticles], d_out, currentNumParticles));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pTorque[sumParticles], d_out, currentNumParticles));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediateTorque[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));
			checkCudaErrors(hipMemcpy(&cpuTest, &intermediateTorque[rbCounter], sizeof(float4), hipMemcpyDeviceToHost));
			if (cpuTest.x != 0 || cpuTest.y != 0 ||cpuTest.z != 0)
			{
//				std::cerr << "Applied torque: (" << cpuTest.x << " " << cpuTest.y << " " << cpuTest.z << ")" << std::endl;
				if (cpuTest.x != cpuTest.x || cpuTest.y != cpuTest.y || cpuTest.z != cpuTest.z)
					*toExit = true;
			}
			hipMemset(d_out, 0, particlesPerObjectThrown[num]);

			CustomMax4RB    max_op;
			float4 init = make_float4(-inf, -inf, -inf, -inf);

			//rigid body positions reduction
			checkCudaErrors(hipcub::DeviceReduce::Reduce(d_temp_storage,
					temp_storage_bytes,
					&pPositions[sumParticles],
					d_out,
					currentNumParticles,
					max_op,
					init));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Reduce(d_temp_storage,
					temp_storage_bytes,
					&pPositions[sumParticles],
					d_out,
					currentNumParticles,
					max_op,
					init));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&intermediatePositions[rbCounter], &d_out[0], sizeof(float4), hipMemcpyDeviceToDevice));


			int *collisionCounter;
			checkCudaErrors(hipMalloc((void**)&collisionCounter, sizeof(int) * currentNumParticles));
			//rigid body torque reduction
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pCountARCollions[sumParticles], collisionCounter, currentNumParticles));
			checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
			checkCudaErrors(hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, &pCountARCollions[sumParticles], collisionCounter, currentNumParticles));

			//copy result to intermediate variable
			checkCudaErrors(hipMemcpy(&ARCollisionsRigidBody[rbCounter], &collisionCounter[0], sizeof(int), hipMemcpyDeviceToDevice));


			checkCudaErrors(hipFree(collisionCounter));
			checkCudaErrors(hipFree(d_out));
			checkCudaErrors(hipFree(d_temp_storage));
			rbCounter++; //increase counter to keep track of rigid bodies processed


		}
		sumParticles += currentNumParticles;
	}

	dim3 blockDim(numThreads, 1);
	dim3 gridDim((numRigidBodies + numThreads - 1) / numThreads, 1);
	if (gridDim.x < 1)
			gridDim.x = 1;
	if(numRigidBodies)
	{

		combineIntermediateResultsAR<< < gridDim, blockDim >> >(
				rbForce, //Output: rigid body forces - one element per rigid body
				rbTorque, //Output: rigid body torques - one element per rigid body
				rbPositions, //Output: rigid body positions - one element per rigid body
				intermediateForce, //Input: reduced rigid body forces - one element per rigid body
				intermediateTorque, //Input: reduced rigid body torques - one element per rigid body
				intermediatePositions, //Input: reduced rigid body positions - one element per rigid body
				ARCollisionsRigidBody, //Input: reduced number of collisions
				numRigidBodies); //Auxil.: number of rigid bodies
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	blockDim = dim3(numThreads, 1);
	gridDim = dim3((numParticles + numThreads - 1) / numThreads, 1);
	if (gridDim.x < 1)
		gridDim.x = 1;
	if(numParticles)
	{
		resetBlockVariablesAR<< < gridDim, blockDim >> >(
				pForce, //Input: rigid body forces - one element per particle
				pTorque, //Input: rigid body torques - one element per particle
				pPositions, //Input: rigid body positions - one element per particle
				pCountARCollions, //Input: AR collisions - one element per particle
				numParticles);
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	hipFree(ARCollisionsRigidBody);
	hipFree(intermediateForce);
	hipFree(intermediateTorque);
	hipFree(intermediatePositions);
}
